#include "hip/hip_runtime.h"
#include <opencv2/core.hpp>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void resizeImageKernel(const uchar3* input, uchar3* output, int inputWidth, int inputHeight, int outputWidth, int outputHeight) {
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;

   if (x < outputWidth && y < outputHeight) {
      int srcX = static_cast<int>(x * static_cast<double>(inputWidth) / outputWidth);
      int srcY = static_cast<int>(y * static_cast<double>(inputHeight) / outputHeight);

      output[y * outputWidth + x] = input[srcY * inputWidth + srcX];
   }
}

extern "C" void resizeImage(const cv::Mat & input, cv::Mat & output, int newWidth, int newHeight) {
   uchar3* d_input;
   uchar3* d_output;

   size_t inputSize = input.rows * input.cols * sizeof(uchar3);
   size_t outputSize = newHeight * newWidth * sizeof(uchar3);

   hipMalloc(&d_input, inputSize);
   hipMalloc(&d_output, outputSize);

   hipMemcpy(d_input, input.data, inputSize, hipMemcpyHostToDevice);

   dim3 blockSize(16, 16);
   dim3 gridSize((newWidth + blockSize.x - 1) / blockSize.x, (newHeight + blockSize.y - 1) / blockSize.y);

   resizeImageKernel << <gridSize, blockSize >> > (d_input, d_output, input.cols, input.rows, newWidth, newHeight);

   hipMemcpy(output.data, d_output, outputSize, hipMemcpyDeviceToHost);

   hipFree(d_input);
   hipFree(d_output);
}

